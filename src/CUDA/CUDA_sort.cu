#include "hip/hip_runtime.h"
/*
 * Cuda++	_sort.cu
 *
 *  Created on: 25/nov/2010
 *      Author: lorenzo
 */

#include "CUDA_sort.cuh"

__constant__ int hilb_N[1];
__constant__ int hilb_N_unsortable[1];
__constant__ int hilb_depth[1];
__constant__ float hilb_box_side[1];

/****************************************************************************************
taken by Journal of Computational Physics, Vol. 227, No. 10. (01 May 2008), pp. 5342-5359
****************************************************************************************/

/**
 * swap Hilbert spacing-filling curve vertices
 */
__device__ void vertex_swap(int *v, int *a, int *b, int const mask) {
	// swap bits comprising Hilbert codes in vertex-to-code lookup table
	int const va = (((*v) >> (*a)) & mask);
	int const vb = (((*v) >> (*b)) & mask);
	(*v) = (*v) ^ (va << (*a)) ^ (vb << (*b)) ^ (va << (*b)) ^ (vb << (*a));
	// update code-to-vertex lookup table
	int c = (*b);
	(*b) = (*a);
	(*a) = c;
}

/**
 * map 3-dimensional point to 1-dimensional point on Hilbert space curve
 */
template<typename number4>
__device__ int hilbert_code(number4 *r) {
	//
	// Jun Wang & Jie Shan, Space-Filling Curve Based Point Clouds Index,
	// GeoComputation, 2005
	//

	// Hilbert code for particle
	int hcode = 0;
	// Hilbert code-to-vertex lookup table
	int a = 21;
	int b = 18;
	int c = 12;
	int d = 15;
	int e = 3;
	int f = 0;
	int g = 6;
	int h = 9;
	// Hilbert vertex-to-code lookup table
	int vc = 1U << b ^ 2U << c ^ 3U << d ^ 4U << e ^ 5U << f ^ 6U << g ^ 7U << h;

	int x, y, z, v;
#define MASK ((1 << 3) - 1)

	// 32-bit integer for 3D Hilbert code allows a maximum of 10 levels
	for (int i = 0; i < *hilb_depth; ++i) {
		// determine Hilbert vertex closest to particle
		x = __signbitf(r->x) & 1;
		y = __signbitf(r->y) & 1;
		z = __signbitf(r->z) & 1;
		// lookup Hilbert code
		v = (vc >> (3 * (x + (y << 1) + (z << 2))) & MASK);

		// scale particle coordinates to subcell
		r->x = 2 * r->x - (0.5f - x);
		r->y = 2 * r->y - (0.5f - y);
		r->z = 2 * r->z - (0.5f - z);
		// apply permutation rule according to Hilbert code
		if (v == 0) {
			vertex_swap(&vc, &b, &h, MASK);
			vertex_swap(&vc, &c, &e, MASK);
		} else if (v == 1 || v == 2) {
			vertex_swap(&vc, &c, &g, MASK);
			vertex_swap(&vc, &d, &h, MASK);
		} else if (v == 3 || v == 4) {
			vertex_swap(&vc, &a, &c, MASK);
#ifdef USE_HILBERT_ALT_3D
			vertex_swap(&vc, &b, &d, MASK);
			vertex_swap(&vc, &e, &g, MASK);
#endif
			vertex_swap(&vc, &f, &h, MASK);
		} else if (v == 5 || v == 6) {
			vertex_swap(&vc, &a, &e, MASK);
			vertex_swap(&vc, &b, &f, MASK);
		} else if (v == 7) {
			vertex_swap(&vc, &a, &g, MASK);
			vertex_swap(&vc, &d, &f, MASK);
		}

		// add vertex code to partial Hilbert code
		hcode = (hcode << 3) + v;
	}
#undef MASK
	return hcode;
}

template<typename number4>
__global__ void hilbert_curve(const number4 *pos, int *hindex) {
	if(IND >= hilb_N[0]) return;
	//
	// We need to avoid ambiguities during the assignment of a particle
	// to a subcell, i.e. the particle position should never lie on an
	// edge or corner of multiple subcells, or the algorithm will have
	// trouble converging to a definite Hilbert curve.
	//
	// Therefore, we use a simple cubic lattice of predefined dimensions
	// according to the number of cells at the deepest recursion level,
	// and round the particle position to the nearest center of a cell.
	//

	number4 r = pos[IND];
	// Hilbert cells per dimension at deepest recursion level
	const int n = 1UL << *hilb_depth;
	// fractional index of particle's Hilbert cell in [0, n)
	r.x /= hilb_box_side[0];
	r.y /= hilb_box_side[0];
	r.z /= hilb_box_side[0];
	r.x = (r.x - floorf(r.x)) * n;
	r.y = (r.y - floorf(r.y)) * n;
	r.z = (r.z - floorf(r.z)) * n;

	// round particle position to center of cell in unit coordinates
	r.x = (floorf(r.x) + 0.5f) / n;
	r.y = (floorf(r.y) + 0.5f) / n;
	r.z = (floorf(r.z) + 0.5f) / n;

	// use symmetric coordinates
	r.x -= 0.5f;
	r.y -= 0.5f;
	r.z -= 0.5f;

	// compute Hilbert code for particle
	const int code = (IND < hilb_N_unsortable[0]) ? IND : hilbert_code<number4>(&r) + hilb_N_unsortable[0];
	hindex[IND] = code;
}

template<typename number, typename number4> 
__global__ void permute_particles(int *sorted_hindex, int *inv_sorted_hindex, number4 *poss, number4 *vels, number4 *Ls,
		GPU_quat<number> *orientations, LR_bonds *bonds, number4 *buff_poss, number4 *buff_vels,
		number4 *buff_Ls, GPU_quat<number> *buff_orientations, LR_bonds *buff_bonds) {
	if(IND >= hilb_N[0]) return;

	const int j = sorted_hindex[IND];

	LR_bonds b = bonds[j];
	LR_bonds buff_b = {P_INVALID, P_INVALID};
	if(b.n3 != P_INVALID) buff_b.n3 = inv_sorted_hindex[b.n3];
	if(b.n5 != P_INVALID) buff_b.n5 = inv_sorted_hindex[b.n5];

	buff_bonds[IND] = buff_b;
	buff_orientations[IND] = orientations[j];
	buff_poss[IND] = poss[j];
	buff_vels[IND] = vels[j];
	buff_Ls[IND] = Ls[j];
}

template<typename number, typename number4>
__global__ void permute_particles(int *sorted_hindex, number4 *poss, number4 *vels, number4 *buff_poss, number4 *buff_vels) {
	if(IND >= hilb_N[0]) return;

	const int j = sorted_hindex[IND];
	buff_poss[IND] = poss[j];
	buff_vels[IND] = vels[j];
}

template<typename number, typename number4>
__global__ void permute_particles(int *sorted_hindex, number4 *poss, number4 *buff_poss) {
	if(IND >= hilb_N[0]) return;

	const int j = sorted_hindex[IND];
	buff_poss[IND] = poss[j];
}

__global__ void get_inverted_sorted_hindex(int *sorted_hindex, int *inv_sorted_hindex) {
	if(IND >= hilb_N[0]) return;

	inv_sorted_hindex[sorted_hindex[IND]] = IND;
}

__global__ void reset_sorted_hindex(int *sorted_hindex) {
	if(IND >= hilb_N[0]) return;

	sorted_hindex[IND] = IND;
}

void init_hilb_symbols(int N, int N_unsortable, int depth, float box_side) {
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(hilb_N), &N, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(hilb_depth), &depth, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(hilb_box_side), &box_side, sizeof(float)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(hilb_N_unsortable), &N_unsortable, sizeof(int)) );
}

template
__global__ void permute_particles<float, float4>(int *sorted_hindex, int *inv_sorted_hindex, float4 *poss, float4 *vels,	float4 *Ls, GPU_quat<float> *orientations, LR_bonds *bonds, float4 *buff_poss, float4 *buff_vels, float4 *buff_Ls, GPU_quat<float> *buff_orientations, LR_bonds *buff_bonds);
template 

__global__ void permute_particles<double, LR_double4>(int *sorted_hindex, int *inv_sorted_hindex, LR_double4 *poss, LR_double4 *vels, LR_double4 *Ls, GPU_quat<double> *orientations, LR_bonds *bonds, LR_double4 *buff_poss, LR_double4 *buff_vels, LR_double4 *buff_Ls, GPU_quat<double> *buff_orientations, LR_bonds *buff_bonds);
template
__global__ void permute_particles<float, float4>(int *sorted_hindex, float4 *poss, float4 *vels, float4 *buff_poss, float4 *buff_vels);
template 
__global__ void permute_particles<double, LR_double4>(int *sorted_hindex, LR_double4 *poss, LR_double4 *vels, LR_double4 *buff_poss, LR_double4 *buff_vels);
template 
__global__ void permute_particles<float, float4>(int *sorted_hindex, float4 *poss, float4 *buff_poss);
template 
__global__ void permute_particles<double, LR_double4>(int *sorted_hindex, LR_double4 *poss, LR_double4 *buff_poss);
template 
__global__ void hilbert_curve<float4>(const float4 *pos, int *hindex);
template 
__global__ void hilbert_curve<LR_double4>(const LR_double4 *pos, int *hindex);
