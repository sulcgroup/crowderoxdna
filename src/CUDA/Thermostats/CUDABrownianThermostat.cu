#include "hip/hip_runtime.h"
/*
 * CUDABrownianThermostat.cpp
 *
 *  Created on: Feb 15, 2013
 *      Author: rovigatti
 */

#include <hiprand/hiprand_kernel.h>

#include "CUDABrownianThermostat.h"

template<typename number, typename number4>
__global__ void brownian_thermostat(hiprandState *rand_state, number4 *vels, number4 *Ls, number rescale_factor, number pt, number pr, int N) {
	if(IND < N) {
		hiprandState state = rand_state[IND];

		if(hiprand_uniform(&state) < pt) {
			number4 v;
			number trash;

			gaussian(state, v.x, v.y);
			gaussian(state, v.z, trash);

			v.x *= rescale_factor;
			v.y *= rescale_factor;
			v.z *= rescale_factor;
			v.w = (v.x*v.x + v.y*v.y + v.z*v.z) * (number) 0.5f;

			vels[IND] = v;
		}

		if(hiprand_uniform(&state) < pr) {
			number4 L;
			number trash;

			gaussian(state, L.x, L.y);
			gaussian(state, L.z, trash);

			L.x *= rescale_factor;
			L.y *= rescale_factor;
			L.z *= rescale_factor;
			L.w = (L.x*L.x + L.y*L.y + L.z*L.z) * (number) 0.5f;

			Ls[IND] = L;
		}

		rand_state[IND] = state;
	}
}

template<typename number, typename number4>
CUDABrownianThermostat<number, number4>::CUDABrownianThermostat() : CUDABaseThermostat<number, number4>(), BrownianThermostat<number>() {

}

template<typename number, typename number4>
CUDABrownianThermostat<number, number4>::~CUDABrownianThermostat() {

}

template<typename number, typename number4>
void CUDABrownianThermostat<number, number4>::get_settings(input_file &inp) {
	BrownianThermostat<number>::get_settings(inp);
	CUDABaseThermostat<number, number4>::get_cuda_settings(inp);
}

template<typename number, typename number4>
void CUDABrownianThermostat<number, number4>::init(int N) {
	BrownianThermostat<number>::init(N);

	this->_setup_rand(N);
}

template<typename number, typename number4>
bool CUDABrownianThermostat<number, number4>::would_activate(llint curr_step) {
	return (curr_step % this->_newtonian_steps == 0);
}

template<typename number, typename number4>
void CUDABrownianThermostat<number, number4>::apply_cuda(number4 *d_poss, GPU_quat<number> *d_orientations, number4 *d_vels, number4 *d_Ls, llint curr_step) {
	if(!would_activate(curr_step)) return;

	brownian_thermostat<number, number4>
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(this->_d_rand_state, d_vels, d_Ls, this->_rescale_factor, this->_pt, this->_pr, this->_N_part);
}

template class CUDABrownianThermostat<float, float4>;
template class CUDABrownianThermostat<double, LR_double4>;
