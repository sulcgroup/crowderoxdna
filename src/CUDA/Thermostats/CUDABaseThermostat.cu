#include "hip/hip_runtime.h"
/*
 * CUDABaseThermostat.cu
 *
 *  Created on: Feb 15, 2013
 *      Author: rovigatti
 */

#include "CUDABaseThermostat.h"

template<typename number, typename number4>
CUDABaseThermostat<number, number4>::~CUDABaseThermostat() {
	CUDA_SAFE_CALL( hipFree(_d_rand_state) );
}
template<typename number, typename number4>
void CUDABaseThermostat<number, number4>::get_cuda_settings(input_file &inp) {
	_launch_cfg.threads_per_block = 64;
	getInputInt(&inp, "threads_per_block", &_launch_cfg.threads_per_block, 0);
}

template<typename number, typename number4>
void CUDABaseThermostat<number, number4>::_setup_rand(int N) {
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<hiprandState>(&_d_rand_state, N * sizeof(hiprandState)) );

	_launch_cfg.blocks.x = N / _launch_cfg.threads_per_block + ((N % _launch_cfg.threads_per_block == 0) ? 0 : 1);
	if(_launch_cfg.blocks.x == 0) _launch_cfg.blocks.x = 1;
	_launch_cfg.blocks.y = _launch_cfg.blocks.z = 1;

	setup_curand<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>(_d_rand_state, _seed, N);
}

template class CUDABaseThermostat<float, float4>;
template class CUDABaseThermostat<double, LR_double4>;

