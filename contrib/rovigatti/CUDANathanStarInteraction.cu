#include "hip/hip_runtime.h"
/*
 * CUDANathanStarInteraction.cu
 *
 *  Created on: 22/feb/2013
 *      Author: lorenzo
 */

#include "CUDANathanStarInteraction.h"

#include "../Lists/CUDASimpleVerletList.h"
#include "../Lists/CUDANoList.h"

/* BEGIN CUDA */

/* System constants */
__constant__ int MD_N[1];
__constant__ float MD_box_side[1];
__constant__ float MD_sqr_patchy_rcut[1];
__constant__ float MD_sqr_patchy_star_rcut[1];
__constant__ float MD_sqr_star_rcut[1];

// patch-patch constants
__constant__ int MD_rep_power[1];
__constant__ float MD_rep_E_cut[1];

__constant__ float MD_patch_angular_cutoff[1];
__constant__ int MD_patch_half_power[1];
__constant__ float MD_patch_pow_sigma[1];
__constant__ float MD_patch_pow_alpha[1];

// star-star constants
__constant__ float  MD_star_f3_2[1];
__constant__ float  MD_star_f1_2[1];
__constant__ float  MD_T[1];
__constant__ float  MD_star_sigma_s[1];
__constant__ float  MD_sqr_star_sigma_s[1];
__constant__ float MD_star_factor[1];

// patchy-star variables
__constant__ int MD_interp_size[1];
__constant__ float MD_xmin[1];
__constant__ float MD_xmax[1];
__constant__ float MD_bin[1];
texture<float, 1, hipReadModeElementType> tex_patchy_star;

#include "../cuda_utils/CUDA_lr_common.cuh"

template <typename number, typename number4>
__device__ number4 minimum_image(number4 &r_i, number4 &r_j) {
	number dx = r_j.x - r_i.x;
	number dy = r_j.y - r_i.y;
	number dz = r_j.z - r_i.z;

	dx -= floorf(dx/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dy -= floorf(dy/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dz -= floorf(dz/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];

	return make_number4<number, number4>(dx, dy, dz, (number) 0.f);
}

template <typename number>
__device__ number interpolate_patchy_star(number x) {
	if(x <= MD_xmin[0]) return 100.;
	if(x >= MD_xmax[0]) return 0.;

	int last = (MD_interp_size[0] - 1);

	int bin_1 = (x - MD_xmin[0]) / MD_bin[0];
	int bin_0 = (bin_1 == 0) ? bin_1 : bin_1 - 1;
	int bin_2 = (bin_1 == last) ? 0 : bin_1 + 1;
	int bin_3 = (bin_2 == last) ? bin_2 : bin_2 + 1;

	number p0 = tex1Dfetch(tex_patchy_star, bin_0);
	number p1 = tex1Dfetch(tex_patchy_star, bin_1);
	number p2 = tex1Dfetch(tex_patchy_star, bin_2);
	number p3 = tex1Dfetch(tex_patchy_star, bin_3);

	number dx = x - bin_1*MD_bin[0] - MD_xmin[0];
	number fx0 = p1;
	number fx1 = p2;
	number derx0 = (p2 - p0) / (2.f*MD_bin[0]);
	number derx1 = (p3 - p1) / (2.f*MD_bin[0]);
	derx0 = derx1 = 0.;

	number D = (2.f*(fx0 - fx1) + (derx0 + derx1)*MD_bin[0]) / MD_bin[0];
	number C = (fx1 - fx0 + (-derx0 - D)*MD_bin[0]);

	number fx = fx0 + dx*(derx0 + dx*(C + dx*D) / SQR(MD_bin[0]));

//	printf("%lf %lf %lf %lf %lf %lf\n", x, fx, p0, p1, p2, p3);
	return fx;
}

template <typename number, typename number4>
__device__ void _patchy_patchy_interaction(number4 &ppos, number4 &qpos, number4 &p_axis, number4 &q_axis, number4 &F, number4 &torque) {
	number4 r = minimum_image<number, number4>(ppos, qpos);
	number sqr_r = CUDA_DOT(r, r);
	if(sqr_r >= MD_sqr_patchy_rcut[0]) return;

	// here everything is done as in Allen's paper
	number rmod = sqrtf(sqr_r);
	number4 r_versor = r / (-rmod);

	// repulsion
	number rep_part = 1.f / powf(sqr_r, MD_rep_power[0] / 2.f);
	number4 force = r_versor * (MD_rep_power[0] * rep_part / rmod);
	F += force;

	number cospr = -CUDA_DOT(p_axis, r_versor);
	if(cospr < 0.) {
		p_axis = -p_axis;
		cospr = -cospr;
	}
	number cosqr = CUDA_DOT(q_axis, r_versor);
	if(cosqr < 0.) {
		q_axis = -q_axis;
		cosqr = -cosqr;
	}
	if(cospr < MD_patch_angular_cutoff[0] || cosqr < MD_patch_angular_cutoff[0]) return;

	// powf generates nan if called with a negative first argument so we have to use the SQR(...) macro and the half exponent
	number cospr_base = powf(SQR(cospr - 1.f), MD_patch_half_power[0] - 1) * (cospr - 1.f);
	// we do this so that later we don't have to divide this number by (cospr - 1), which could be 0
	number cospr_part = cospr_base * (cospr - 1.f);
	number p_mod = expf(-cospr_part / (2.f*MD_patch_pow_sigma[0]));

	number cosqr_base = powf(SQR(cosqr - 1.f), MD_patch_half_power[0] - 1) * (cosqr - 1.f);
	number cosqr_part = cosqr_base * (cosqr - 1.f);
	number q_mod = expf(-cosqr_part / (2.f*MD_patch_pow_sigma[0]));

	number sqr_surf_dist = SQR(rmod - 1.f);
	number r8b10 = SQR(SQR(sqr_surf_dist)) / MD_patch_pow_alpha[0];
	number exp_part = -1.001f*expf(-0.5f*r8b10*sqr_surf_dist);

	// radial part
	number4 tmp_force = r_versor * (p_mod*q_mod * 5.f*(rmod - 1.f)*exp_part*r8b10);

	// angular p part
	number der_p = exp_part * q_mod * (MD_patch_half_power[0]*p_mod * cospr_base / MD_patch_pow_sigma[0]);
	number4 p_ortho = p_axis + cospr*r_versor;
	tmp_force -= p_ortho * (der_p/rmod);

	// angular q part
	number der_q = exp_part * p_mod * (-MD_patch_half_power[0]*q_mod * cosqr_base / MD_patch_pow_sigma[0]);
	number4 q_ortho = q_axis - cosqr*r_versor;
	tmp_force -= q_ortho * (der_q/rmod);

//	printf("%d %lf %lf %lf %lf %lf\n", IND, p_mod, q_mod, cospr_part, cosqr_part, sqrt(CUDA_DOT(tmp_force, tmp_force)));

	F += tmp_force;
	torque += _cross<number, number4>(r_versor, p_axis) * der_p;
}

template <typename number, typename number4>
__device__ void _patchy_star_interaction(number4 &ppos, number4 &qpos, number4 &F) {
	number4 r = minimum_image<number, number4>(ppos, qpos);
	number sqr_r = CUDA_DOT(r, r);
	if(sqr_r >= MD_sqr_patchy_star_rcut[0]) return;

	number mod_r = sqrt(sqr_r);
	number force_module = interpolate_patchy_star<number>(mod_r);

	F.x -= r.x * (force_module/mod_r);
	F.y -= r.y * (force_module/mod_r);
	F.z -= r.z * (force_module/mod_r);
}

template <typename number, typename number4>
__device__ void _star_star_interaction(number4 &ppos, number4 &qpos, number4 &F) {
	number4 r = minimum_image<number, number4>(ppos, qpos);
	number sqr_r = CUDA_DOT(r, r);
	if(sqr_r >= MD_sqr_star_rcut[0]) return;

	number mod_r = sqrt(sqr_r);

	number common_fact = MD_star_factor[0] * 5.f * MD_T[0] * MD_star_f3_2[0] / 18.f;

	if(sqr_r < MD_sqr_star_sigma_s[0]) {
		// force over r
		number force_mod = common_fact / sqr_r;
		F -= r * force_mod;
	}
	else {
		number exp_factor = expf(-(mod_r - MD_star_sigma_s[0])*MD_star_f1_2[0]/(2.*MD_star_sigma_s[0]));

		number i_f = 1.f / (1.f + MD_star_f1_2[0]*0.5f);
		// force over r
		number force_mod = common_fact * i_f * exp_factor * (MD_star_sigma_s[0]/(sqr_r*mod_r) + MD_star_f1_2[0]/(2.f*sqr_r));
		F -= r * force_mod;
	}
}

// forces + second step without lists
template <typename number, typename number4>
__global__ void NS_forces(number4 *poss, GPU_quat<number> *orientations, number4 *forces, number4 *torques) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	number4 T = make_number4<number, number4>(0, 0, 0, 0);
	number4 ppos = poss[IND];
	GPU_quat<number> po = orientations[IND];
	number4 a1, a2, a3, b1, b2, b3;
	get_vectors_from_quat<number,number4>(po, a1, a2, a3);

	int ptype = get_particle_type<number, number4>(ppos);

	for(int j = 0; j < MD_N[0]; j++) {
		if(j != IND) {
			number4 qpos = poss[j];
			int qtype = get_particle_type<number, number4>(qpos);

			int type = ptype + qtype;
			if(type == NathanStarInteraction<number>::PATCHY_PATCHY) {
				GPU_quat<number> qo = orientations[j];
				get_vectors_from_quat<number,number4>(qo, b1, b2, b3);
				_patchy_patchy_interaction<number, number4>(ppos, qpos, a3, b3, F, T);
			}
			else if(type == NathanStarInteraction<number>::PATCHY_POLYMER) _patchy_star_interaction<number, number4>(ppos, qpos, F);
			else _star_star_interaction<number, number4>(ppos, qpos, F);
		}
	}

	forces[IND] = F;
	torques[IND] = _vectors_transpose_number4_product(a1, a2, a3, T);
}

//Forces + second step with verlet lists
template <typename number, typename number4>
__global__ void NS_forces(number4 *poss, GPU_quat<number> *orientations, number4 *forces, number4 *torques, int *matrix_neighs, int *number_neighs) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	number4 T = make_number4<number, number4>(0, 0, 0, 0);
	number4 ppos = poss[IND];
	GPU_quat<number> po = orientations[IND];
	number4 a1, a2, a3, b1, b2, b3;
	get_vectors_from_quat<number,number4>(po, a1, a2, a3);

	int ptype = get_particle_type<number, number4>(ppos);

	int num_neighs = number_neighs[IND];
	for(int j = 0; j < num_neighs; j++) {
		int k_index = matrix_neighs[j*MD_N[0] + IND];

		number4 qpos = poss[k_index];
		int qtype = get_particle_type<number, number4>(qpos);

		int type = ptype + qtype;
		if(type == NathanStarInteraction<number>::PATCHY_PATCHY) {
			GPU_quat<number> qo = orientations[k_index];
			get_vectors_from_quat<number,number4>(qo, b1, b2, b3);
			_patchy_patchy_interaction<number, number4>(ppos, qpos, a3, b3, F, T);
		}
		else if(type == NathanStarInteraction<number>::PATCHY_POLYMER) _patchy_star_interaction<number, number4>(ppos, qpos, F);
		else _star_star_interaction<number, number4>(ppos, qpos, F);
	}

	forces[IND] = F;
	torques[IND] = _vectors_transpose_number4_product(a1, a2, a3, T);
}

/* END CUDA PART */

template<typename number, typename number4>
CUDANathanStarInteraction<number, number4>::CUDANathanStarInteraction() : CUDABaseInteraction<number, number4>(), NathanStarInteraction<number>() {
	_d_patchy_star = NULL;
}

template<typename number, typename number4>
CUDANathanStarInteraction<number, number4>::~CUDANathanStarInteraction() {
	if(_d_patchy_star != NULL) CUDA_SAFE_CALL( hipFree(_d_patchy_star) );
}

template<typename number, typename number4>
void CUDANathanStarInteraction<number, number4>::get_settings(input_file &inp) {
	NathanStarInteraction<number>::get_settings(inp);
}

template<typename number, typename number4>
void CUDANathanStarInteraction<number, number4>::_setup_cuda_interp() {
	int size = this->_spl_patchy_star->size;
	float *fx = new float[size];
	for(int i = 0; i < size; i++) fx[i] = -gsl_spline_eval_deriv(this->_spl_patchy_star, this->_spl_patchy_star->x[i], this->_acc_patchy_star);

	int v_size = size*sizeof(float);
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<float>(&_d_patchy_star, v_size) );
	CUDA_SAFE_CALL( hipMemcpy(_d_patchy_star, fx, size*sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipBindTexture(NULL, tex_patchy_star, _d_patchy_star, v_size) );

	delete[] fx;

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_interp_size), &size, sizeof(int)) );
	float f_copy = this->_spl_patchy_star->interp->xmin;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_xmin), &f_copy, sizeof(float)) );
	f_copy = this->_spl_patchy_star->interp->xmax;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_xmax), &f_copy, sizeof(float)) );
	f_copy = this->_spl_patchy_star->x[1] - this->_spl_patchy_star->x[0];
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_bin), &f_copy, sizeof(float)) );
}

template<typename number, typename number4>
void CUDANathanStarInteraction<number, number4>::cuda_init(number box_side, int N) {
	CUDABaseInteraction<number, number4>::cuda_init(box_side, N);
	NathanStarInteraction<number>::init();

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_rep_power), &this->_rep_power, sizeof(int)) );
	int patch_half_power = this->_patch_power / 2;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_half_power), &patch_half_power, sizeof(int)) );

	float f_copy = box_side;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_box_side), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_patchy_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_patchy_rcut), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_patchy_star_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_patchy_star_rcut), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_star_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_star_rcut), &f_copy, sizeof(float)) );
	f_copy = this->_rep_E_cut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_rep_E_cut), &f_copy, sizeof(float)) );
	f_copy = this->_patch_angular_cutoff;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_angular_cutoff), &f_copy, sizeof(float)) );
	f_copy = this->_patch_pow_alpha;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_pow_alpha), &f_copy, sizeof(float)) );
	f_copy = this->_patch_pow_sigma;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_pow_sigma), &f_copy, sizeof(float)) );

	f_copy = this->_T;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_T), &f_copy, sizeof(float)) );
	f_copy = this->_star_f1_2;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_star_f1_2), &f_copy, sizeof(float)) );
	f_copy = this->_star_f3_2;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_star_f3_2), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_star_sigma_s;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_star_sigma_s), &f_copy, sizeof(float)) );
	f_copy = this->_star_sigma_s;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_star_sigma_s), &f_copy, sizeof(float)) );
	f_copy = this->_star_factor;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_star_factor), &f_copy, sizeof(float)) );

	_setup_cuda_interp();
}

template<typename number, typename number4>
void CUDANathanStarInteraction<number, number4>::compute_forces(CUDABaseList<number, number4> *lists, number4 *d_poss, GPU_quat<number> *d_orientations, number4 *d_forces, number4 *d_torques, LR_bonds *d_bonds) {
	CUDASimpleVerletList<number, number4> *_v_lists = dynamic_cast<CUDASimpleVerletList<number, number4> *>(lists);
	if(_v_lists != NULL) {
		if(_v_lists->use_edge()) throw oxDNAException("use_edge unsupported by NathanStarInteraction");
		else {
			NS_forces<number, number4>
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_orientations, d_forces, d_torques, _v_lists->_d_matrix_neighs, _v_lists->_d_number_neighs);
			CUT_CHECK_ERROR("forces_second_step FS simple_lists error");
		}
	}

	CUDANoList<number, number4> *_no_lists = dynamic_cast<CUDANoList<number, number4> *>(lists);
	if(_no_lists != NULL) {
		NS_forces<number, number4>
			<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
			(d_poss, d_orientations, d_forces, d_torques);
		CUT_CHECK_ERROR("forces_second_step FS no_lists error");
	}
}

template class CUDANathanStarInteraction<float, float4>;
template class CUDANathanStarInteraction<double, LR_double4>;
