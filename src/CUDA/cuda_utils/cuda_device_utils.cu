#include <stdio.h>

#include "cuda_device_utils.h"

int get_device_count() {
	int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount FAILED, CUDA Driver and Runtime CUDA Driver and Runtime version may be mismatched, exiting.\n");
		exit(-1);
	}

	return deviceCount;
}

void check_device_existance(int device) {
	if(device >= get_device_count()) {
		fprintf(stderr, "The selected device doesn't exist, exiting.\n");
		exit(-1);
	}
}

hipDeviceProp_t get_current_device_prop() {
	int curr_dev;
	hipGetDevice(&curr_dev);
	return get_device_prop(curr_dev);
}

hipDeviceProp_t get_device_prop(int device) {
	check_device_existance(device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	return deviceProp;
}

hipError_t set_device(int device) {
	check_device_existance(device);
	hipDeviceReset();
	return hipSetDevice(device);
}
