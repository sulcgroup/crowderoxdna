#include "hip/hip_runtime.h"
/*
 * CUDAStarrInteraction.cu
 *
 *  Created on: 22/feb/2013
 *      Author: lorenzo
 */

#include "CUDAStarrInteraction.h"

#include "../Lists/CUDASimpleVerletList.h"
#include "../Lists/CUDANoList.h"
#include "../Particles/CustomParticle.h"

/* CUDA constants */
__constant__ bool MD_starr_model[1];
__constant__ int MD_mode[1];
__constant__ int MD_N[1];
__constant__ int MD_N_hubs[1];
__constant__ int MD_N_per_strand[1];
__constant__ int MD_N_per_tetramer[1];
__constant__ float MD_box_side[1];

__constant__ float MD_LJ_sigma[3];
__constant__ float MD_LJ_sqr_sigma[3];
__constant__ float MD_LJ_rcut[3];
__constant__ float MD_LJ_sqr_rcut[3];
__constant__ float MD_LJ_E_cut[3];
__constant__ float MD_der_LJ_E_cut[3];
__constant__ float MD_fene_K[1];
__constant__ float MD_fene_sqr_r0[1];
__constant__ float MD_lin_k[1];
__constant__ float MD_sqr_rcut[1];

#include "../cuda_utils/CUDA_lr_common.cuh"

template <typename number, typename number4>
__device__ number4 minimum_image(const number4 &r_i, const number4 &r_j) {
	number dx = r_j.x - r_i.x;
	number dy = r_j.y - r_i.y;
	number dz = r_j.z - r_i.z;

	dx -= floorf(dx/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dy -= floorf(dy/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dz -= floorf(dz/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];

	return make_number4<number, number4>(dx, dy, dz, (number) 0.f);
}

template <typename number, typename number4>
__device__ number quad_minimum_image_dist(const number4 &r_i, const number4 &r_j) {
	number dx = r_j.x - r_i.x;
	number dy = r_j.y - r_i.y;
	number dz = r_j.z - r_i.z;

	dx -= floorf(dx/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dy -= floorf(dy/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dz -= floorf(dz/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];

	return dx*dx + dy*dy + dz*dz;
}

template <typename number, typename number4>
__device__ void _two_body(number4 &r, int pbtype, int qbtype, int p_idx, int q_idx, number4 &F, bool disable_pairing) {
	int ptype = (pbtype == N_DUMMY || pbtype == P_HUB) ? 0 : 1;
	int qtype = (qbtype == N_DUMMY || qbtype == P_HUB) ? 0 : 1;
	int int_type = ptype + qtype;

	int int_btype = pbtype + qbtype;
	if(int_type == 2 && (int_btype != 3 || disable_pairing)) int_type = 1;

	number sqr_r = CUDA_DOT(r, r);
	number mod_r = sqrt(sqr_r);
	number sqr_sigma_r = MD_LJ_sqr_sigma[int_type] / sqr_r;
	number part = sqr_sigma_r*sqr_sigma_r*sqr_sigma_r;
	number force_mod = 24.f*part*(2.f*part - 1.f) / sqr_r + MD_der_LJ_E_cut[int_type]/mod_r;
	number energy = 4.f*part*(part - 1.f) - MD_LJ_E_cut[int_type] - (mod_r - MD_LJ_rcut[int_type])*MD_der_LJ_E_cut[int_type];

	if(sqr_r > MD_LJ_sqr_rcut[int_type]) energy = force_mod = 0.f;
//	else printf("%d %d %f %f %d\n", p_idx, q_idx, energy, force_mod, int_type);

	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy*0.5f;
}

template<typename number, typename number4>
__device__ void _fene(number4 &r, number4 &F) {
	number sqr_r = CUDA_DOT(r, r);
	number energy = -0.5f*MD_fene_K[0]*MD_fene_sqr_r0[0]*logf(1.f - sqr_r/MD_fene_sqr_r0[0]);
	// this number is the module of the force over r, so we don't have to divide the distance
	// vector by its module
	number force_mod = -MD_fene_K[0]*MD_fene_sqr_r0[0] / (MD_fene_sqr_r0[0] - sqr_r);
//	printf("%d %lf\n", IND, force_mod);

	F.x -= r.x * force_mod;
	F.y -= r.y * force_mod;
	F.z -= r.z * force_mod;
	F.w += energy*0.5f;
}

template <typename number, typename number4>
__device__ void _particle_particle_bonded_interaction(number4 &ppos, number4 &qpos, number4 &F, bool only_fene=false) {
	int pbtype = get_particle_btype<number, number4>(ppos);
	int p_idx = get_particle_index<number, number4>(ppos);
	int qbtype = get_particle_btype<number, number4>(qpos);
	int q_idx = get_particle_index<number, number4>(qpos);

	number4 r = minimum_image<number, number4>(ppos, qpos);
	if(!only_fene) _two_body<number, number4>(r, pbtype, qbtype, p_idx, q_idx, F, true);
	_fene<number, number4>(r, F);
}

template <typename number, typename number4>
__device__ void _particle_particle_interaction(number4 &ppos, number4 &qpos, number4 &F, int *strand_ids) {
	int pbtype = get_particle_btype<number, number4>(ppos);
	int p_idx = get_particle_index<number, number4>(ppos);
	int qbtype = get_particle_btype<number, number4>(qpos);
	int q_idx = get_particle_index<number, number4>(qpos);

	bool same_strand = (strand_ids[p_idx] == strand_ids[q_idx]);
	bool neighbours = (abs(p_idx - q_idx) == 2);

	number4 r = minimum_image<number, number4>(ppos, qpos);
	_two_body<number, number4>(r, pbtype, qbtype, p_idx, q_idx, F, same_strand && neighbours);
}

template <typename number, typename number4>
__device__ void _particle_all_bonded_interactions(number4 &ppos, LR_bonds &bs, number4 &F, number4 *poss, number4 *forces) {
	int pbtype = get_particle_btype<number, number4>(ppos);
	// backbone or hub
	if(pbtype == N_DUMMY || pbtype == P_HUB) {
		bool has_n3 = (bs.n3 != P_INVALID);
		bool has_n5 = (bs.n5 != P_INVALID);
		// backbone
		if(has_n3) {
			number4 qpos = poss[bs.n3];
			_particle_particle_bonded_interaction<number, number4>(ppos, qpos, F);
		}

		// backbone-base
		if(pbtype == N_DUMMY) {
			number4 qpos = poss[IND + 1];
			_particle_particle_bonded_interaction<number, number4>(ppos, qpos, F, true);
		}

		if(has_n5) {
			number4 qpos = poss[bs.n5];
			_particle_particle_bonded_interaction<number, number4>(ppos, qpos, F);
		}
	}
	// base
	else {
		// base-backbone
		number4 qpos = poss[IND - 1];
		_particle_particle_bonded_interaction<number, number4>(ppos, qpos, F, true);
	}
}

// forces + second step without lists
template <typename number, typename number4>
__global__ void Starr_forces(number4 *poss, number4 *forces, LR_bonds *bonds, int *strand_ids) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	LR_bonds bs = bonds[IND];
	number4 ppos = poss[IND];

	_particle_all_bonded_interactions<number, number4>(ppos, bs, F, poss, forces);

	for(int j = 0; j < MD_N[0]; j++) {
		if(j != IND && bs.n3 != j && bs.n5 != j) {
			number4 qpos = poss[j];
			_particle_particle_interaction<number, number4>(ppos, qpos, F, strand_ids);
		}
	}

	forces[IND] = F;
}

// forces + second step with verlet lists
template <typename number, typename number4>
__global__ void Starr_forces(number4 *poss, number4 *forces, int *matrix_neighs, int *number_neighs, LR_bonds *bonds, int *strand_ids) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	LR_bonds bs = bonds[IND];
	number4 ppos = poss[IND];

	_particle_all_bonded_interactions<number, number4>(ppos, bs, F, poss, forces);

	const int num_neighs = number_neighs[IND];
	for(int j = 0; j < num_neighs; j++) {
		const int k_index = matrix_neighs[j*MD_N[0] + IND];

		number4 qpos = poss[k_index];
		_particle_particle_interaction<number, number4>(ppos, qpos, F, strand_ids);
	}

	forces[IND] = F;
}

template<typename number, typename number4>
__device__ void _three_body(number4 &ppos, LR_bonds &bs, number4 &F, number4 *poss, number4 *n3_forces, number4 *n5_forces) {
	if(bs.n3 == P_INVALID || bs.n5 == P_INVALID) return;

	number4 n3_pos = poss[bs.n3];
	number4 n5_pos = poss[bs.n5];

	number4 dist_pn3 = minimum_image<number, number4>(ppos, n3_pos);
	number4 dist_pn5 = minimum_image<number, number4>(n5_pos, ppos);

	number sqr_dist_pn3 = CUDA_DOT(dist_pn3, dist_pn3);
	number sqr_dist_pn5 = CUDA_DOT(dist_pn5, dist_pn5);
	number i_pn3_pn5 = 1.f / sqrtf(sqr_dist_pn3*sqr_dist_pn5);
	number cost = CUDA_DOT(dist_pn3, dist_pn5) * i_pn3_pn5;

	number cost_n3 = cost / sqr_dist_pn3;
	number cost_n5 = cost / sqr_dist_pn5;
	number force_mod_n3 = i_pn3_pn5 + cost_n3;
	number force_mod_n5 = i_pn3_pn5 + cost_n5;

	F += dist_pn3*(force_mod_n3*MD_lin_k[0]) - dist_pn5*(force_mod_n5*MD_lin_k[0]);
	F.w += MD_lin_k[0]*(1.f - cost);

	number4 n3_force = dist_pn5*(i_pn3_pn5*MD_lin_k[0]) - dist_pn3*(cost_n3*MD_lin_k[0]);
	number4 n5_force = dist_pn5*(cost_n5*MD_lin_k[0]) - dist_pn3*(i_pn3_pn5*MD_lin_k[0]);
	/*n3_forces[bs.n3] = n3_force;
	  n5_forces[bs.n5] = n5_force;*/
	LR_atomicAddXYZ(n3_forces + bs.n3, n3_force);
	LR_atomicAddXYZ(n5_forces + bs.n5, n5_force);
}

template <typename number, typename number4>
__global__ void three_body_forces(number4 *poss, number4 *forces, number4 *n3_forces, number4 *n5_forces, LR_bonds *bonds) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND];
	LR_bonds bs = bonds[IND];
	number4 ppos = poss[IND];
	int btype = get_particle_btype<number, number4>(ppos);

	if(MD_starr_model[0] && btype == P_HUB) {
	  /*int base_idx = IND - (IND % MD_N_per_tetramer[0]);
		for(int i = 0; i < MD_N_per_tetramer[0]; i += MD_N_per_strand[0]) {
			bs.n3 = base_idx + i;
			if(bs.n3 != IND) {
				_three_body<number, number4>(ppos, bs, F, poss, n3_forces, n5_forces);
			}
		}*/
	}
	else _three_body<number, number4>(ppos, bs, F, poss, n3_forces, n5_forces);

	forces[IND] = F;
}

template <typename number, typename number4>
__global__ void sum_three_body(number4 *forces, number4 *n3_forces, number4 *n5_forces) {
	if(IND >= MD_N[0]) return;

	number4 F = forces[IND] + n3_forces[IND] + n5_forces[IND];
	forces[IND] = F;
}

template <typename number, typename number4>
__global__ void hub_forces(number4 *poss, number4 *forces, number4 *n3_forces, number4 *n5_forces, int *hubs, hub_bonds *bonds, LR_bonds *n3n5) {
	if(IND >= MD_N_hubs[0]) return;

	int idx_hub = hubs[IND];
	hub_bonds hub_bonds = bonds[IND];
	number4 pos_hub = poss[idx_hub];
	number4 F = forces[idx_hub];
	LR_bonds bs_hub = n3n5[idx_hub];

	for(int an = 0; an < (HUB_SIZE-1); an++) {
		int bonded_neigh = hub_bonds.n[an];
		// since bonded neighbours of hub are in the hub's neighbouring list, the LJ interaction between
		// the two, from the point of view of the hub, has been already computed and hence the hub-particle
		// interaction reduces to just the fene
		if(bonded_neigh != P_INVALID) {
			_particle_particle_bonded_interaction<number, number4>(pos_hub, poss[bonded_neigh], F, true);
			if(MD_starr_model[0]) {
				bs_hub.n3 = bonded_neigh;
				_three_body<number, number4>(pos_hub, bs_hub, F, poss, n3_forces, n5_forces);
			}
		}
	}

	forces[idx_hub] = F;
}

template<typename number, typename number4>
CUDAStarrInteraction<number, number4>::CUDAStarrInteraction() {
	_N_hubs = -1;
	_d_hubs = _d_strand_ids = NULL;
	_d_hub_neighs = NULL;
	_d_n3_forces = _d_n5_forces = NULL;
}

template<typename number, typename number4>
CUDAStarrInteraction<number, number4>::~CUDAStarrInteraction() {
	if(_d_strand_ids != NULL) {
		CUDA_SAFE_CALL( hipFree(_d_strand_ids) );
	}

	if(_d_hubs != NULL) {
		CUDA_SAFE_CALL( hipFree(_d_hubs) );
		CUDA_SAFE_CALL( hipFree(_d_hub_neighs) );
	}

	if(_d_n3_forces == NULL) {
		CUDA_SAFE_CALL( hipFree(_d_n3_forces) );
		CUDA_SAFE_CALL( hipFree(_d_n5_forces) );
	}
}

template<typename number, typename number4>
void CUDAStarrInteraction<number, number4>::get_settings(input_file &inp) {
	StarrInteraction<number>::get_settings(inp);

	int sort_every;
	if(getInputInt(&inp, "CUDA_sort_every", &sort_every, 0) == KEY_FOUND) {
		if(sort_every > 0) throw oxDNAException("Starr interaction is not compatible with particle sorting, aborting");
	}
}

template<typename number, typename number4>
void CUDAStarrInteraction<number, number4>::cuda_init(number box_side, int N) {
	CUDABaseInteraction<number, number4>::cuda_init(box_side, N);
	StarrInteraction<number>::init();

	if(this->_mode != StarrInteraction<number>::STRANDS) _setup_hubs();
	_setup_strand_ids();

	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_n3_forces, this->_N*sizeof(number4)) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_n5_forces, this->_N*sizeof(number4)) );
	CUDA_SAFE_CALL( hipMemset(_d_n3_forces, 0, this->_N*sizeof(number4)) );
	CUDA_SAFE_CALL( hipMemset(_d_n5_forces, 0, this->_N*sizeof(number4)) );

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_starr_model), &this->_starr_model, sizeof(bool)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_mode), &this->_mode, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N_per_strand), &this->_N_per_strand, sizeof(int)) );
	float f_copy = box_side;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_box_side), &f_copy, sizeof(float)) );
	f_copy = this->_lin_k;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_lin_k), &f_copy, sizeof(float)) );
	f_copy = this->_fene_K;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_fene_K), &f_copy, sizeof(float)) );
	f_copy = this->_fene_sqr_r0;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_fene_sqr_r0), &f_copy, sizeof(float)) );
	f_copy = this->_sqr_rcut;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_sqr_rcut), &f_copy, sizeof(float)) );

	COPY_ARRAY_TO_CONSTANT(MD_LJ_sigma, this->_LJ_sigma, 3);
	COPY_ARRAY_TO_CONSTANT(MD_LJ_sqr_sigma, this->_LJ_sqr_sigma, 3);
	COPY_ARRAY_TO_CONSTANT(MD_LJ_rcut, this->_LJ_rcut, 3);
	COPY_ARRAY_TO_CONSTANT(MD_LJ_sqr_rcut, this->_LJ_sqr_rcut, 3);
	COPY_ARRAY_TO_CONSTANT(MD_LJ_E_cut, this->_LJ_E_cut, 3);
	COPY_ARRAY_TO_CONSTANT(MD_der_LJ_E_cut, this->_der_LJ_E_cut, 3);
}

template<typename number, typename number4>
void CUDAStarrInteraction<number, number4>::_setup_strand_ids() {
	BaseParticle<number> **particles = new BaseParticle<number> *[this->_N];
	StarrInteraction<number>::allocate_particles(particles, this->_N);
	int N_strands;
	StarrInteraction<number>::read_topology(this->_N, &N_strands, particles);

	int *h_strand_ids = new int[this->_N];

	for(int i = 0; i < this->_N; i++) h_strand_ids[i] = particles[i]->strand_id;
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_strand_ids, this->_N*sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpy(_d_strand_ids, h_strand_ids, this->_N*sizeof(int), hipMemcpyHostToDevice) );

	delete[] h_strand_ids;

	for(int i = 0; i < this->_N; i++) delete particles[i];
	delete[] particles;
}

template<typename number, typename number4>
void CUDAStarrInteraction<number, number4>::_setup_hubs() {	
	BaseParticle<number> **particles = new BaseParticle<number> *[this->_N];
	StarrInteraction<number>::allocate_particles(particles, this->_N);
	int N_strands;
	StarrInteraction<number>::read_topology(this->_N, &N_strands, particles);

	int N_per_tetramer = 4*this->_N_per_strand;
	N_per_tetramer = 68;

	_N_hubs = this->_N_tetramers*4 + this->_N_dimers*2;
	int *h_hubs = new int[_N_hubs];
	hub_bonds *h_hub_neighs = new hub_bonds[_N_hubs];

	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_hubs, _N_hubs*sizeof(int)) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<hub_bonds>(&_d_hub_neighs, _N_hubs*sizeof(hub_bonds)) );

	int rel_idx_hub = 0;
	for(int i = 0; i < this->_N; i++) {
		CustomParticle<number> *p = static_cast<CustomParticle<number> *>(particles[i]);
		if(p->btype == P_HUB) {
			h_hubs[rel_idx_hub] = p->index;

			// now load all the hub_bonds structures by looping over all the bonded neighbours
			int nn = 0;
			for(typename set<CustomParticle<number> *>::iterator it = p->bonded_neighs.begin(); it != p->bonded_neighs.end(); it++) {
				if((*it) != p->n5) {
					h_hub_neighs[rel_idx_hub].n[nn] = (*it)->index;
					nn++;
				}
			}
			for(; nn < HUB_SIZE-1; nn++) h_hub_neighs[rel_idx_hub].n[nn] = P_INVALID;
			rel_idx_hub++;
		}
	}

	if(rel_idx_hub != _N_hubs) throw oxDNAException("%d hubs found, should have been %d", rel_idx_hub, _N_hubs);

	CUDA_SAFE_CALL( hipMemcpy(_d_hubs, h_hubs, _N_hubs*sizeof(int), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(_d_hub_neighs, h_hub_neighs, _N_hubs*sizeof(hub_bonds), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N_hubs), &_N_hubs, sizeof(int)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N_per_tetramer), &N_per_tetramer, sizeof(int)) );

	for(int i = 0; i < this->_N; i++) delete particles[i];
	delete[] particles;
	delete[] h_hubs;
	delete[] h_hub_neighs;
}

template<typename number, typename number4>
void CUDAStarrInteraction<number, number4>::compute_forces(CUDABaseList<number, number4> *lists, number4 *d_poss, GPU_quat<number> *d_orientations, number4 *d_forces, number4 *d_torques, LR_bonds *d_bonds) {
	three_body_forces<number, number4>
		<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
		(d_poss, d_forces, _d_n3_forces, _d_n5_forces, d_bonds);
	CUT_CHECK_ERROR("three_body_forces error");

	if(this->_mode != StarrInteraction<number>::STRANDS) {
		hub_forces<number, number4>
			<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
			(d_poss, d_forces, _d_n3_forces, _d_n5_forces, _d_hubs, _d_hub_neighs, d_bonds);
		CUT_CHECK_ERROR("hub_forces error");
	}

	sum_three_body<number, number4>
		<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
		(d_forces, _d_n3_forces, _d_n5_forces);
	CUT_CHECK_ERROR("sum_three_body error");

	CUDA_SAFE_CALL( hipMemset(_d_n3_forces, 0, this->_N*sizeof(number4)) );
	CUDA_SAFE_CALL( hipMemset(_d_n5_forces, 0, this->_N*sizeof(number4)) );

	CUDASimpleVerletList<number, number4> *_v_lists = dynamic_cast<CUDASimpleVerletList<number, number4> *>(lists);
	if(_v_lists != NULL) {
		if(_v_lists->use_edge()) throw oxDNAException("use_edge unsupported by StarrInteraction");
		else {
			Starr_forces<number, number4>
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss, d_forces, _v_lists->_d_matrix_neighs, _v_lists->_d_number_neighs, d_bonds, _d_strand_ids);
			CUT_CHECK_ERROR("Starr_forces Verlet Lists error");
		}
	}
	else {
		CUDANoList<number, number4> *_no_lists = dynamic_cast<CUDANoList<number, number4> *>(lists);

		if(_no_lists != NULL) {
			Starr_forces<number, number4>
				<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
				(d_poss,  d_forces, d_bonds, _d_strand_ids);
			CUT_CHECK_ERROR("Starr_forces no_lists error");
		}
	}	
}

extern "C" IBaseInteraction<float> *make_CUDAStarrInteraction_float() {
	return new CUDAStarrInteraction<float, float4>();
}

extern "C" IBaseInteraction<double> *make_CUDAStarrInteraction_double() {
	return new CUDAStarrInteraction<double, LR_double4>();
}

template class CUDAStarrInteraction<float, float4>;
template class CUDAStarrInteraction<double, LR_double4>;
