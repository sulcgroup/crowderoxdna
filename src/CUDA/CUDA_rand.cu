#include "hip/hip_runtime.h"
#include "CUDA_rand.cuh"

__global__ void setup_curand(hiprandState *rand_state, const llint seed, const int N) {
	if(IND >= N) return;

	hiprand_init(seed, IND, 0, &rand_state[IND]);
}
