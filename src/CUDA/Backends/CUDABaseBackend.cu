#include "hip/hip_runtime.h"
/*
 * CUDABaseBackend.cpp
 *
 *  Created on: 25/nov/2010
 *      Author: lorenzo
 */

#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include "CUDABaseBackend.h"
#include "../Lists/CUDAListFactory.h"
#include "../Interactions/CUDAInteractionFactory.h"
#include "../../Utilities/oxDNAException.h"

using namespace std;

template<typename number, typename number4>
CUDABaseBackend<number, number4>::CUDABaseBackend() : _device_number(0), _sort_every(0) {
	_particles_kernel_cfg.blocks = dim3(1, 1, 1);
	_particles_kernel_cfg.threads_per_block = 0;
	_particles_kernel_cfg.shared_mem = 0;

	_device_number = -1;
	_sqr_verlet_skin = 0.f;

	_cuda_lists = NULL;
	_cuda_interaction = NULL;
	_d_poss = NULL;
	_d_bonds = NULL;
	_d_orientations = NULL;
	_d_list_poss = NULL;
	_d_are_lists_old = NULL;
	_d_hindex = NULL;
	_d_sorted_hindex = NULL;
	_d_inv_sorted_hindex = NULL;
	_d_buff_poss = NULL;
	_d_buff_bonds = NULL;
	_d_buff_orientations = NULL;
	_h_poss = NULL;
	_h_orientations = NULL;
	_h_bonds = NULL;
}

template<typename number, typename number4>
CUDABaseBackend<number, number4>::~CUDABaseBackend() {
	if (_cuda_lists != NULL) {
		_cuda_lists->clean();
		delete _cuda_lists;
	}
	if (_cuda_interaction != NULL) delete _cuda_interaction;

	if (_d_poss != NULL){
		CUDA_SAFE_CALL( hipFree(_d_poss) );
		CUDA_SAFE_CALL( hipFree(_d_bonds) );
		CUDA_SAFE_CALL( hipFree(_d_orientations) );
		CUDA_SAFE_CALL( hipFree(_d_list_poss) );
		CUDA_SAFE_CALL( hipHostFree(_d_are_lists_old) );
	}

	if(_sort_every > 0) {
		if (_d_hindex != NULL){
			CUDA_SAFE_CALL( hipFree(_d_hindex) );
			CUDA_SAFE_CALL( hipFree(_d_sorted_hindex) );
			CUDA_SAFE_CALL( hipFree(_d_inv_sorted_hindex) );
			CUDA_SAFE_CALL( hipFree(_d_buff_poss) );
			CUDA_SAFE_CALL( hipFree(_d_buff_bonds) );
			CUDA_SAFE_CALL( hipFree(_d_buff_orientations) );
		}
	}

	if (_h_poss != NULL) delete[] _h_poss;
	if (_h_orientations != NULL) delete[] _h_orientations;
	if (_h_bonds != NULL) delete[] _h_bonds;
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_host_to_gpu() {
	CUDA_SAFE_CALL( hipMemcpy(_d_poss, _h_poss, _vec_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(_d_bonds, _h_bonds, _bonds_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(_d_orientations, _h_orientations, _orient_size, hipMemcpyHostToDevice) );
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_gpu_to_host() {
	CUDA_SAFE_CALL( hipMemcpy(_h_poss, _d_poss, _vec_size, hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(_h_bonds, _d_bonds, _bonds_size, hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(_h_orientations, _d_orientations, _orient_size, hipMemcpyDeviceToHost) );
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::get_settings(input_file &inp) {
	if(getInputInt(&inp, "CUDA_device", &_device_number, 0) == KEY_NOT_FOUND) {
		OX_LOG(Logger::LOG_INFO, "CUDA device not specified");
		_device_number = -1;
	}
	else OX_LOG(Logger::LOG_INFO, "Using CUDA device %d", _device_number);

	if(getInputInt(&inp, "CUDA_sort_every", &_sort_every, 0) == KEY_NOT_FOUND)
		OX_LOG(Logger::LOG_INFO, "CUDA sort_every not specified, using 0");

	getInputInt(&inp, "threads_per_block", &_particles_kernel_cfg.threads_per_block, 0);

	float verlet_skin;
	if(getInputFloat(&inp, "verlet_skin", &verlet_skin, 0) == KEY_FOUND) _sqr_verlet_skin = SQR(verlet_skin);

	_cuda_interaction = CUDAInteractionFactory::make_interaction<number, number4>(inp);
	_cuda_interaction->get_settings(inp);
	_cuda_interaction->get_cuda_settings(inp);

	_cuda_lists = CUDAListFactory::make_list<number, number4>(inp);
	_cuda_lists->get_settings(inp);

	// check that the box is cubic
	string my_box;
	if(getInputString(&inp, "box_type", my_box, 0) == KEY_FOUND) if(my_box != "cubic") throw oxDNAException("The CUDA backend only supports cubic boxes");

	string reload_from;
	if(getInputString(&inp, "reload_from", reload_from, 0) == KEY_FOUND) throw oxDNAException("The CUDA backend does not support reloading checkpoints, owing to its intrisincally stochastic nature");
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_choose_device () {
	OX_LOG(Logger::LOG_INFO, "Choosing device automatically");

	int ndev = -1, trydev = 0;
	hipDeviceProp_t tryprop;

	hipGetDeviceCount (&ndev);
	OX_LOG(Logger::LOG_INFO, "Computer has %i devices", ndev);
	while (trydev < ndev) {
		OX_LOG(Logger::LOG_INFO, " - Trying device %i", trydev);
		tryprop = get_device_prop (trydev);
		OX_LOG(Logger::LOG_INFO, " -- device %i has properties %i.%i", trydev, tryprop.major, tryprop.minor);
		if (tryprop.major < 2 && tryprop.minor <= 2) {
			OX_LOG(Logger::LOG_INFO, " -- Device properties are not good. Skipping it", trydev);
			trydev ++;
			continue;
		}
		set_device (trydev);
		int *dummyptr = NULL;
		hipError_t ggg = GpuUtils::LR_cudaMalloc<int> (&dummyptr, (size_t)sizeof(int));
		if(ggg == hipSuccess) {
			OX_LOG(Logger::LOG_INFO, " -- using device %i", trydev);
			hipFree (dummyptr);
			break;
		}
		else {
			OX_LOG(Logger::LOG_INFO, " -- device %i not available ...", trydev);
		}
		trydev ++;
	}

	if (trydev == ndev) throw oxDNAException("No suitable devices available");

	OX_LOG(Logger::LOG_INFO, " --- Running on device %i", trydev);
	_device_prop = get_device_prop(trydev);
	_device_number = trydev;
	// gpu device chosen
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::init_cuda(ConfigInfo<number> &config_info) {
	if(_device_number < 0)	_choose_device();
	set_device(_device_number);
	_device_prop = get_device_prop(_device_number);

	CUDA_SAFE_CALL( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );

	_prv_config_info = config_info;
	number box_side = config_info.box->box_sides().x;
	int N = *config_info.N;

	_cuda_interaction->cuda_init(box_side, N);

	_vec_size = sizeof(number4) * N;
	_orient_size = sizeof(GPU_quat<number>) * N;
	_bonds_size = sizeof(LR_bonds) * N;

	// GPU memory allocations
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_poss, _vec_size) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<LR_bonds>(&_d_bonds, _bonds_size) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<GPU_quat<number>  >(&_d_orientations, _orient_size) );
	CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_list_poss, _vec_size) );
	CUDA_SAFE_CALL( hipHostMalloc(&_d_are_lists_old, sizeof(bool), hipHostMallocDefault) );

	CUDA_SAFE_CALL( hipMemset(_d_list_poss, 0, _vec_size) );

	// CPU memory allocations
	_h_poss = new number4[N];
	_h_orientations = new GPU_quat<number>[N];
	_h_bonds = new LR_bonds[N];

	// setup kernels' configurations
	_init_CUDA_kernel_cfgs();
	_cuda_lists->init(N, box_side, _cuda_interaction->get_cuda_rcut());

	if(_sort_every > 0) {
		int uns = 0;

		// fixed value for depth (8): changing this value does not significantly affect performances
		init_hilb_symbols(N, uns, 8, (float) box_side);

		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_hindex, N*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_sorted_hindex, N*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_inv_sorted_hindex, N*sizeof(int)) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_buff_poss, _vec_size) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<LR_bonds>(&_d_buff_bonds, _bonds_size) );
		CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<GPU_quat<number>  >(&_d_buff_orientations, _orient_size) );

		reset_sorted_hindex
			<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
			(_d_sorted_hindex);
	}
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_init_CUDA_kernel_cfgs() {
	if(_particles_kernel_cfg.threads_per_block == 0) {
		_particles_kernel_cfg.threads_per_block = 2*_device_prop.warpSize;
		OX_LOG(Logger::LOG_INFO, "threads_per_block was not specified or set to 0. The default value (%d) will be used", 2*_device_prop.warpSize);
	}

	int N = *_prv_config_info.N;
	_particles_kernel_cfg.blocks.x = N / _particles_kernel_cfg.threads_per_block + ((N % _particles_kernel_cfg.threads_per_block == 0) ? 0 : 1);
	if(_particles_kernel_cfg.blocks.x == 0) _particles_kernel_cfg.blocks.x = 1;
	_particles_kernel_cfg.blocks.y = _particles_kernel_cfg.blocks.z = 1;

	_cuda_interaction->set_launch_cfg(_particles_kernel_cfg);

	OX_DEBUG("Particle kernel cfg: threads_per_block = %d, blocks = (%d, %d, %d)", _particles_kernel_cfg.threads_per_block,
			_particles_kernel_cfg.blocks.x, _particles_kernel_cfg.blocks.y, _particles_kernel_cfg.blocks.z);
}

template<typename number, typename number4>
void CUDABaseBackend<number, number4>::_sort_index() {
	reset_sorted_hindex
		<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
		(_d_sorted_hindex);
	CUT_CHECK_ERROR("reset_sorted_hindex error");

	hilbert_curve<number4>
		<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
		(_d_poss, _d_hindex);
	CUT_CHECK_ERROR("hilbert_curve error");

	thrust::device_ptr<int> _d_hindex_p(_d_hindex);
	thrust::device_ptr<int> _d_sorted_hindex_p(_d_sorted_hindex);
	// sort d_sorted_hindex by using d_hindex
	thrust::sort_by_key(_d_hindex_p, _d_hindex_p + *_prv_config_info.N, _d_sorted_hindex_p);
	get_inverted_sorted_hindex
		<<<_particles_kernel_cfg.blocks, _particles_kernel_cfg.threads_per_block>>>
		(_d_sorted_hindex, _d_inv_sorted_hindex);
}

// template instantiations
template class CUDABaseBackend<float, float4>;
template class CUDABaseBackend<double, LR_double4>;
